﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"

__device__ void user_program(struct VirtualMemory *vm, uchar *input, uchar *results, int input_size) {
    for (int i = 0; i < input_size; i++) // i: offset
        vm_write(vm, i, input[i]);
	printf("write page fault: %d\n",*(vm->pagefault_num_ptr));
    for (int i = input_size - 1; i >= input_size - 32769; i--) {
        int value = vm_read(vm, i);
    }
	printf("read page fault: %d\n", *(vm->pagefault_num_ptr));

	vm_snapshot(vm, results, 0, input_size);
	printf("snapshot page fault: %d\n", *(vm->pagefault_num_ptr));
}
