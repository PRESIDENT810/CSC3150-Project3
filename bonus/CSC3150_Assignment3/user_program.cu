#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results, int input_size) {
//	for (int i = 0; i < input_size; i++) // i: offset
//		vm_write(vm, i, input[i]);
//	printf("write page fault: %d\n", *(vm->pagefault_num_ptr));
//	for (int i = input_size - 1; i >= input_size - 32769; i--) {
//		int value = vm_read(vm, i);
//	}
//	printf("read page fault: %d\n", *(vm->pagefault_num_ptr));
//
//	vm_snapshot(vm, results, 0, input_size);
//	printf("snapshot page fault: %d\n", *(vm->pagefault_num_ptr));
}


__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results, int input_size, int thread_id) {
//	printf("thread: %d\n", thread_id);
	int order = 0;

	for (int i = 0; i < input_size/4; i++) { // i: offset
		//if (thread_id != vm->current_thread) {
		//	printf("thread id %d, current thread %d, wait\n", thread_id, vm->current_thread);
		//	continue;
		//}

		//if (i % 4 != thread_id) continue;

		int fuck = i + thread_id * (input_size / 4);
		// printf("thread id %d, fuck is %d\n", thread_id, fuck);

		if (thread_id == 0) {
			vm_write(vm, fuck, input[fuck], thread_id);
		}
		__syncthreads();

		if (thread_id == 1) {
			vm_write(vm, fuck, input[fuck], thread_id);
		}
		__syncthreads();

		if (thread_id == 2) {
			vm_write(vm, fuck, input[fuck], thread_id);
		}
		__syncthreads();

		if (thread_id == 3) {
			vm_write(vm, fuck, input[fuck], thread_id);
		}
		__syncthreads();

		// printf("thread id %d, current thread %d, current+1\n", thread_id, vm->current_thread);
		//vm->current_thread = (vm->current_thread+1)%4;

		
//		__syncthreads(); // all four threads have written, synchronize then continue 
	}
	printf("write page fault count: %d\n", *(vm->pagefault_num_ptr));

    for (int i = input_size - 1; i >= input_size - 32769; i--) {
		int fuck = i + thread_id * (input_size / 4);

		if (thread_id == 0) {
			int value = vm_read(vm, fuck, thread_id);
		}
		__syncthreads();

		if (thread_id == 1) {
			int value = vm_read(vm, fuck, thread_id);
		}
		__syncthreads();

		if (thread_id == 2) {
			int value = vm_read(vm, fuck, thread_id);
		}
		__syncthreads();

		if (thread_id == 3) {
			int value = vm_read(vm, fuck, thread_id);
		}
		__syncthreads();

		//vm->current_thread = (vm->current_thread + 1) % 4;
    }
	printf("read page fault count: %d\n", *(vm->pagefault_num_ptr));

	if (thread_id != 0) return;
	vm_snapshot(vm, results, 0, input_size, thread_id);
	printf("snapshot page fault %d\n", *(vm->pagefault_num_ptr));
}
