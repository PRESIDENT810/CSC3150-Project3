﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdlib.h"
#include "stdio.h"

// extern int pagefault_num;

__device__ void init_invert_page_table(VirtualMemory *vm) {

    for (int i = 0; i < 1024; i++) {
        vm->invert_page_table[i] = -1; // invalid := MSB is 1
        vm->invert_page_table[i + 1024] = i%4;
    }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
             int *invert_page_table, int *pagefault_num_ptr,
             int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
             int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
             int PAGE_ENTRIES, int current_thread) {
    // init variables
    vm->buffer = buffer;
    vm->storage = storage;
    vm->invert_page_table = invert_page_table;
    vm->pagefault_num_ptr = pagefault_num_ptr;
	vm->current_thread = current_thread;

    struct Page_node *head;
    struct Page_node *current;
	head = (struct Page_node *) malloc(100);
	head->nxt = NULL;
	//hipMalloc((void **)head, 100);
	current = head;
    for (int i = 0; i < 1024; i++) {
        struct Page_node *temp;
		//hipMalloc((void **)temp, 100);
		temp = (struct Page_node *) malloc(100);
        temp->page_idx = i;
        temp->nxt = NULL;
        current->nxt = temp;
        current = temp;
		current->nxt = NULL;
		//free(temp);
    }

    vm->LRU_head = head;

    // init constants
    vm->PAGESIZE = PAGESIZE;
    vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
    vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
    vm->STORAGE_SIZE = STORAGE_SIZE;
    vm->PAGE_ENTRIES = PAGE_ENTRIES;

    // before first vm_write or vm_read
    init_invert_page_table(vm);
}

__device__ int get_LRUidx(VirtualMemory *vm) { // get the least used index, which is a logical/disk memory address
    return vm->LRU_head->nxt->page_idx;
}


__device__ int search_pageidx(VirtualMemory *vm, int page_num) {
    for (int i = 0; i < 1024; i++) {
        if (vm->invert_page_table[i] == page_num) return i;
    }
    return -1;
}


__device__ void update_stack(VirtualMemory *vm, int idx) {
    struct Page_node *current = vm->LRU_head;

    while (current->nxt->page_idx != idx) current = current->nxt;

    struct Page_node *target = current->nxt;
    current->nxt = target->nxt;

	while (current->nxt != NULL) {
		current = current->nxt;
	}

    current->nxt = target;
    target->nxt = NULL;
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value, int thread_id) {
    /* Complete vm_write function to write value into data buffer */
    int page_num = addr / 32; // addr is the address of disk/logical memory, and page_num is the corresponding page number

	// printf("address %d\n", addr);
	int search_result = search_pageidx(vm, page_num);
	
	if (search_result == -1 || vm->invert_page_table[search_result+1024] != thread_id) { // not found or not this thread
		printf("write page fault %d with thread %d\n",page_num, thread_id);
		(*(vm->pagefault_num_ptr))++;
		int LRU_idx = get_LRUidx(vm); // LRU index is the index of page table instead of the address of logical memory/disk
        int disk_addr = vm->invert_page_table[LRU_idx];
    
		if (disk_addr != -1) {
            for (int i = 0; i < 32; i++) { // swap out
                vm->storage[disk_addr * 32 + i] = vm->buffer[LRU_idx * 32 + i];
            }
        }

		// update page table
		vm->invert_page_table[LRU_idx] = page_num;
		vm->invert_page_table[LRU_idx + 1024] = thread_id;

        for (int i = 0; i < 32; i++) { // swap in
            vm->buffer[LRU_idx * 32 + i] = vm->storage[page_num * 32 + i];
        }

    }

    vm->buffer[search_pageidx(vm, page_num) * 32 + addr % 32] = value; // write into main memory
    update_stack(vm, search_pageidx(vm, page_num));

	return;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr, int thread_id) {
    /* Complate vm_read function to read single element from data buffer */
    /* Complete vm_write function to write value into data buffer */
    int page_num = addr / 32; // addr is the address of disk/logical memory, and page_num is the corresponding page number

	int search_result = search_pageidx(vm, page_num);
	if (search_result == -1 || vm->invert_page_table[search_result + 1024] != thread_id) { // not found or not this thread
		printf("read page fault %d with thread %d\n", page_num, thread_id);
		(*(vm->pagefault_num_ptr))++;
		int LRU_idx = get_LRUidx(vm); // LRU index is the index of page table instead of the address of logical memory/disk
		int disk_addr = vm->invert_page_table[LRU_idx];

		if (disk_addr != -1) {
			for (int i = 0; i < 32; i++) { // swap out
				vm->storage[disk_addr * 32 + i] = vm->buffer[LRU_idx * 32 + i];
			}
		}

		// update page table
		vm->invert_page_table[LRU_idx] = page_num; 
		vm->invert_page_table[LRU_idx + 1024] = thread_id;

		for (int i = 0; i < 32; i++) { // swap in
			vm->buffer[LRU_idx * 32 + i] = vm->storage[page_num * 32 + i];
		}

	}

    uchar content = vm->buffer[search_pageidx(vm, page_num) * 32 + addr % 32]; // read character
    update_stack(vm, search_pageidx(vm, page_num));

    return content;
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset, int input_size, int thread_id) {
    /* Complete snapshot function togther with vm_read to load elements from data to result buffer */
	int temp = 0;
    for (int i=0; i<input_size;i++){
        int value = vm_read(vm,i, thread_id);
        results[i+offset] = value;
    }
}

